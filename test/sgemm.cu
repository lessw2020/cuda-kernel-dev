#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <utils.cuh>

#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Please select a kernel (range 0 - 11, here 0 is for NVIDIA cuBLAS).\n");
        exit(EXIT_FAILURE);
    }

    // cuda kernel num
    int kernel_num = atoi(argv[1]);
    if (kernel_num < 0 || kernel_num > 11) {
        printf("Please enter a valid kernel number (0-11).\n");
        exit(EXIT_FAILURE);
    } else {
        printf("Select kernel %d.\n", kernel_num);
    };

    // Declare handle, create handle, hipblasCreate will return a value of type hipblasStatus_t,
    // which is used to determine if the handle was created successfully (value is 0)
    hipblasHandle_t handle;
    if (hipblasCreate(&handle)) {
        printf("Create cublas handle error.\n");
        exit(EXIT_FAILURE);
    };

    // Use cudaEvent for GPU stream timing, cudaEvent is equivalent to publishing event tasks in the target stream
    float elapsed_time;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);

    // matrix size
    int size_len = 24;
    int SIZE[size_len];
    for (int i = 0; i < size_len; i++)
        SIZE[i] = 256 * (i + 1);

    int m, n, k, max_size;
    max_size = SIZE[size_len - 1];
    printf("max_size=%d\n", max_size);

    float alpha = 1.0, beta = 0.; //two arbitrary input parameters, C=α*AB+β*C

    float *A = NULL, *B = NULL, *C = NULL, *C_ref = NULL;     //host matrices
    float *dA = NULL, *dB = NULL, *dC = NULL, *dC_ref = NULL; //device matrices

    A = (float *) malloc(sizeof(float) * max_size * max_size);
    B = (float *) malloc(sizeof(float) * max_size * max_size);
    C = (float *) malloc(sizeof(float) * max_size * max_size);
    C_ref = (float *) malloc(sizeof(float) * max_size * max_size);

    randomize_matrix(A, max_size * max_size);
    randomize_matrix(B, max_size * max_size);
    randomize_matrix(C, max_size * max_size);
    copy_matrix(C, C_ref, max_size * max_size);

    cudaCheck(hipMalloc((void **) &dA, sizeof(float) * max_size * max_size));
    cudaCheck(hipMalloc((void **) &dB, sizeof(float) * max_size * max_size));
    cudaCheck(hipMalloc((void **) &dC, sizeof(float) * max_size * max_size));
    cudaCheck(hipMalloc((void **) &dC_ref, sizeof(float) * max_size * max_size));

    cudaCheck(hipMemcpy(dA, A, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dB, B, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC, C, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC_ref, C_ref, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));

    int repeat_times = 10;
    for (int i = 0; i < size_len; i++) {
        m = n = k = SIZE[i];

        printf("m=n=k=%d\n", m);
        // Verify computation correctness, and pre-execute once before kernel timing to avoid cold start error
        if (kernel_num != 0) {
            test_kernel(0, m, n, k, alpha, dA, dB, beta, dC_ref, handle);      // cuBLAS
            test_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC, handle); // user define
            hipDeviceSynchronize();
            hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost);
            hipMemcpy(C_ref, dC_ref, sizeof(float) * m * n, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();

            if (!verify_matrix(C_ref, C, m * n)) {
                printf("Failed to pass the correctness verification against NVIDIA cuBLAS. Exited.\n");
                exit(EXIT_FAILURE);
            }
        }
        hipDeviceSynchronize();

        hipEventRecord(beg);
        for (int j = 0; j < repeat_times; j++) {
            test_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC, handle);
        }
        hipEventRecord(end);
        hipEventSynchronize(beg);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time, beg, end);
        elapsed_time /= 1000.; //convert to seconds

        printf("Average elapsed time: (%f) second, performance: (%f) GFLOPS. size: (%d).\n",
               elapsed_time / repeat_times, 2. * 1e-9 * repeat_times * m * n * k / elapsed_time, m);
        fflush(stdout);
        copy_matrix(C_ref, C, m * n); //sync C with cuBLAS to prepare for the next run
    }

    // Free CPU and GPU memory
    free(A);
    free(B);
    free(C);
    free(C_ref);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipFree(dC_ref);

    return 0;
};
